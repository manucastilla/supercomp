// g++ -DTHRUST_DEVICE_SYSTEM=THRUST_DEVICE_SYSTEM_OMP -I../../../thrust/ -fopenmp -x c++ exemplo2.cu -o exemplo2 && ./exemplo2 < ../17-intro-gpu/stocks2.csv
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <math.h>
#include <thrust/iterator/constant_iterator.h>

int main()
{
    int N = 0;
    thrust::host_vector<double> hostMicrosoft;
    thrust::host_vector<double> hostApple;

    double a, m;
    while (std::cin.fail() == false)
    {
        N += 1;

        std::cin >> a;
        std::cin >> m;
        hostMicrosoft.push_back(m);
        hostApple.push_back(a);
    }

    thrust::device_vector<double> diferenca(N);
    thrust::device_vector<double> MSFT(hostMicrosoft);
    thrust::device_vector<double> AAPL(hostApple);
    thrust::device_vector<double> mean_vector(N);
    thrust::device_vector<double> var(N);
    thrust::device_vector<double> var_double(N);

    // diference
    thrust::transform(MSFT.begin(), MSFT.end(), AAPL.begin(), diferenca.begin(), thrust::minus<double>());

    // mean
    double mean = thrust::reduce(diferenca.begin(), diferenca.end(), 0, thrust::plus<double>()) / N;

    // thrust::fill(mean_vector.begin(), mean_vector.end(), mean);

    thrust::transform(diferenca.begin(), diferenca.end(), thrust::constant_iterator<double>(mean), var.begin(), thrust::minus<double>());

    thrust::transform(var.begin(), var.end(), var.begin(),
                      var_double.begin(), thrust::multiplies<double>());

    // for (auto i = var_double.begin(); i != var_double.end(); i++)
    // {
    //     std::cout << *i / N << " "; // este acesso é rápido -- CPU
    // }
    double variancia = thrust::reduce(var_double.begin(), var_double.end(), 0, thrust::plus<double>()) / N;
    std::cout << "variancia: " << variancia << "\n";
}