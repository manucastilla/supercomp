// g++ -DTHRUST_DEVICE_SYSTEM=THRUST_DEVICE_SYSTEM_OMP -I../../../thrust/ -fopenmp -x c++ exemplo3.cu -o exemplo3 && ./exemplo3 < ../17-intro-gpu/stocks.txt
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <thrust/count.h>

struct is_positivo
{
    __host__ __device__ bool operator()(const double &x)
    {
        return x > 0;
    }
};

struct is_negative
{
    __host__ __device__ bool operator()(const double &x)
    {
        return x < 0;
    }
};

int main()
{
    int N = 0;
    thrust::host_vector<double> vec_cpu;

    while (std::cin.fail() == false)
    {
        N += 1;

        double price;
        std::cin >> price;
        vec_cpu.push_back(price);
    }

    // std::cout << N << "\n";

    thrust::device_vector<double>
        stocks(vec_cpu);

    thrust::device_vector<double> ganho_diario(N - 1);

    thrust::transform(stocks.begin() + 1, stocks.end(), stocks.begin(), ganho_diario.begin(), thrust::minus<double>());

    // quantas vezes o valor subiu?
    int result = thrust::count_if(ganho_diario.begin(), ganho_diario.end(), is_positivo());
    std::cout << result << "\n";

    // qual é o aumento médio, considerando só as vezes em que o valor aumentou de fato?
    thrust::replace_if(ganho_diario.begin(), ganho_diario.end(), ganho_diario.begin(), is_negative(), 0);

    double positiveSoma = thrust::reduce(ganho_diario.begin(), ganho_diario.end(), 0.0, thrust::plus<double>());

    double aumento_medio = positiveSoma / result;
    std::cout << aumento_medio << "\n";

    // for (auto i = stocks.begin(); i != stocks.end(); i++)
    // {
    //     std::cout << *i << " "; // este acesso é rápido -- CPU
    // }
    // printf("\n");

    // for (auto i = ganho_diario.begin(); i != ganho_diario.end(); i++)
    // {
    //     std::cout << *i << " "; // este acesso é rápido -- CPU
    // }
    // printf("\n");
}