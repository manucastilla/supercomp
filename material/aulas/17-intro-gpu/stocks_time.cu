//  g++ -DTHRUST_DEVICE_SYSTEM=THRUST_DEVICE_SYSTEM_OMP -I../../../thrust/ -fopenmp -x c++ stocks_time.cu -o stocks_time-cpu && ./stocks_time-cpu < stocks.txt
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <chrono>

int main()
{
    int N = 0;
    thrust::host_vector<double> vec_cpu;

    while (std::cin.fail() == false)
    {
        N += 1;
        double price;
        std::cin >> price;
        vec_cpu.push_back(price);
    }

    std::cout << N << "\n";

    // alocado na CPU

    // aloca vetor na GPU e transfere dados CPU->GPU
    std::chrono::time_point<std::chrono::system_clock> start, end, start_c, end_c;
    start = std::chrono::system_clock::now();

    thrust::device_vector<double>
        vec_gpu(vec_cpu);

    end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = end - start;

    std::cout << "elapsed time da alocação e cópia: " << elapsed_seconds.count() << "s\n";
    //processa vec_gpu

    // ver os tempos separados
    // apenas alocação
    //thrust::device_vector<double>
    //   vec_gpu(sizeof(vec_cpu));
    //apenas cópia
    // vec_gpu = ve= cpu;
}