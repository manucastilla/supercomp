#include "hip/hip_runtime.h"
//  g++ -DTHRUST_DEVICE_SYSTEM=THRUST_DEVICE_SYSTEM_OMP -I../../../thrust/ -fopenmp -x c++ stocks_reduction.cu -o stocks_reduction-cpu && ./stocks_reduction-cpu < stocks.txt
// https://thrust.github.io/doc/group__reductions.html
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <thrust/reduce.h>
#include <chrono>

int main()
{
    int N = 0;
    thrust::host_vector<double> vec_cpu;

    while (std::cin.fail() == false)
    {
        N += 1;

        double price;
        std::cin >> price;
        vec_cpu.push_back(price);
    }

    // std::cout << N << "\n";

    thrust::device_vector<double>
        vec_gpu(vec_cpu);

    // porque eu vou querer transferir do host vector para o device vector para fazer as coisas de reduce?

    // O preço médio das ações nos últimos 10 anos.
    double mean = thrust::reduce(vec_gpu.begin(), vec_gpu.end(), 0, thrust::plus<double>()) / N;
    std::cout << "preço médio no último 10 anos: " << mean << "\n";

    //O preço médio das ações no último ano (365 dias atrás).
    double mean_year = thrust::reduce(vec_gpu.begin() + N - 365, vec_gpu.end(), 0, thrust::plus<double>()) / N;
    std::cout << "maior valor do último ano: " << mean_year << "\n";

    // maior valor da sequência inteira
    double max = thrust::reduce(vec_gpu.begin(), vec_gpu.end(), 0, thrust::maximum<double>());
    std::cout << "maior valor da sequência inteira: " << max << "\n";

    // menor valor da sequência inteira
    double min = thrust::reduce(vec_gpu.begin(), vec_gpu.end(), max, thrust::minimum<double>());
    std::cout << "menor valor da sequência inteira: " << min << "\n";

    // maior valor do último ano
    double max_year = thrust::reduce(vec_gpu.begin() + N - 365, vec_gpu.end(), 0, thrust::maximum<double>());
    std::cout << "maior valor do último ano: " << max_year << "\n";

    // menor valor do último ano
    double min_year = thrust::reduce(vec_gpu.begin() + N - 365, vec_gpu.end(), max_year, thrust::minimum<double>());
    std::cout << "menor valor do último ano: " << min_year << "\n";
}