//g++ -DTHRUST_DEVICE_SYSTEM=THRUST_DEVICE_SYSTEM_OMP -I../../../thrust/ -fopenmp -x c++ stock-apple-micro.cu -o stock-apple-micro-cpu && ./stock-apple-micro-cpu < stocks2.csv

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>

int main()
{
    int N = 0;
    thrust::host_vector<double> hostMicrosoft;
    thrust::host_vector<double> hostApple;

    double a, m;
    while (std::cin.fail() == false)
    {
        N += 1;

        std::cin >> a;
        std::cin >> m;
        hostMicrosoft.push_back(m);
        hostApple.push_back(a);
    }

    thrust::device_vector<double> diferenca(N);
    thrust::device_vector<double> MSFT(hostMicrosoft);
    thrust::device_vector<double> AAPL(hostApple);

    // diference
    thrust::transform(MSFT.begin(), MSFT.end(), AAPL.begin(), diferenca.begin(), thrust::minus<double>());

    // mean
    double mean = thrust::reduce(diferenca.begin(), diferenca.end(), 0, thrust::plus<double>()) / N;
    std::cout << "diferença média entre os preços das ações AAPL e MSFT: " << mean << "\n";
}