//  g++ -DTHRUST_DEVICE_SYSTEM=THRUST_DEVICE_SYSTEM_OMP -I../../../thrust/ -fopenmp -x c++ exemplo2-transform.cu -o exemplo2-transform-cpu && ./exemplo2-transform-cpu

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <iostream>
#include <thrust/iterator/constant_iterator.h>

int main()
{
    thrust::device_vector<double> V1(10, 0);
    thrust::sequence(V1.begin(), V1.end());

    thrust::device_vector<double> V2(10, 0);
    thrust::fill(V2.begin(), V2.begin() + 5, 5.5);
    thrust::fill(V2.begin() + 5, V2.end(), 10);

    thrust::device_vector<double> V3(10);
    thrust::device_vector<double> V4(10);

    thrust::transform(V1.begin(), V1.end(), V2.begin(), V3.begin(), thrust::plus<double>());
    thrust::transform(V1.begin(), V1.end(), thrust::constant_iterator<double>(0.5), V4.begin(), thrust::multiplies<double>());

    printf("V1: ");
    for (thrust::device_vector<double>::iterator i = V1.begin(); i != V1.end(); i++)
    {
        std::cout << *i << " ";
    }
    printf("\n");

    printf("V2: ");
    for (thrust::device_vector<double>::iterator i = V2.begin(); i != V2.end(); i++)
    {
        std::cout << *i << " ";
    }
    printf("\n");

    printf("V3: ");
    for (thrust::device_vector<double>::iterator i = V3.begin(); i != V3.end(); i++)
    {
        std::cout << *i << " ";
    }
    printf("\n");

    printf("V4: ");
    for (thrust::device_vector<double>::iterator i = V4.begin(); i != V4.end(); i++)
    {
        std::cout << *i << " ";
    }
    printf("\n");
}
