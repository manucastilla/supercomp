//  g++ -DTHRUST_DEVICE_SYSTEM=THRUST_DEVICE_SYSTEM_OMP -I../../../thrust/ -fopenmp -x c++ exemplo1-criacao-iteracao.cu -o exemplo1-criacao-iteracao-cpu && ./exemplo1-criacao-iteracao-cpu

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>

int main()
{
    thrust::host_vector<double> host(5, 0);
    host[4] = 35;

    /* na linha abaixo os dados são copiados
       para GPU */
    thrust::device_vector<double> dev(host);
    /* a linha abaixo só muda o vetor na CPU */
    host[2] = 12;

    printf("Host vector: ");
    for (auto i = host.begin(); i != host.end(); i++)
    {
        std::cout << *i << " "; // este acesso é rápido -- CPU
    }
    // i iterator:
    // host.begin() te ddar o iterador (aponta para o primeiro elemento)
    // para pegar o conteúdo do iterador para que oq está sendo apontado
    // isso serve o *
    // HOST - cpu: tua máquina, aonde programa , totalmente alocado na máquina da rápida

    printf("\n");

    printf("Device vector: ");
    // cada vez que faz *i vai buscar na gpu e fazer na cpu (mostrar na tela) tem que copiar, por isso
    // que o processo é lento.
    for (auto i = dev.begin(); i != dev.end(); i++)
    {
        std::cout << *i << " "; // este acesso é lento! -- GPU
    }
    printf("\n");
    // DEVICE - gpu: fica mais longe
}
