#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_real_distribution.h>

// nvcc -arch=sm_70 -std=c++14 exemplo6_disc.cu -o exemplo6_disc && ./exemplo6_disc


struct raw_access {
    thrust::minstd_rand rng;
    thrust::uniform_real_distribution<double> dist;

    raw_access (thrust::uniform_real_distribution<double> dist, thrust::minstd_rand rng) : dist(dist), rng(rng) {};

    __device__ __host__
    double operator()(const int &i) {
        rng.discard(i * 2);
        
        double x = dist(rng);
        double y = dist(rng);
        double power = pow(x, 2) + pow(y, 2);
        if (power <= 1)
        {
            return 1.0;
        }

        return 0.0;
        
    }
};

// http://www.cplusplus.com/reference/random/linear_congruential_engine/discard/
int main(){

    int N = 10000;
    thrust::minstd_rand rng(100);
    thrust::uniform_real_distribution<double> dist(0.0, 1.0);

    thrust::device_vector<double> vetor(N);

    thrust::counting_iterator<int> iter(0);
    raw_access ra(dist, rng);
    thrust::transform(iter, iter+vetor.size(), vetor.begin(), ra);

    double sum = thrust::reduce(vetor.begin(), vetor.end(), 0.0, thrust::plus<double>());

    double pi = (double)4 * sum / N;
    std::cout << "pi de monte carlo " << pi; 
    printf("\n");
}
